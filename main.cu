
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    hipDeviceSynchronize();

    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_out, sizeof(float) * N);
    hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);


    // Main function
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    std::cout << "I am editing from visual studio code!!!" << std::endl;
    std::cout << out[0] << std::endl;
}